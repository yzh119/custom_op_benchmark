#include "hip/hip_runtime.h"
/* TODOs
 * - segment_reduce_forward, segment_reduce_backward;
 * - switch backend from aten to dlpack
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Type.h>
#include <c10/util/Exception.h>
#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define AT_CASE_ITYPE(enum_type, type, DTYPE, NAME, ...)                    \
  case enum_type: {                                                         \
    const at::Type& dtype = DTYPE;                                          \
    using idx_t = type;                                                     \
    switch (dtype.scalarType()) {                                           \
      case at::ScalarType::Float: {                                         \
        using data_t = float;                                               \
        return __VA_ARGS__();                                               \
      }                                                                     \
      case at::ScalarType::Double: {                                        \
        using data_t = double;                                              \
        return __VA_ARGS__();                                               \
      }                                                                     \
      default:                                                              \
        AT_ERROR(#NAME, " not implemented for '", dtype.toString(), "'");   \
    }                                                                       \
  }                

#define AT_DISPATCH_IDX_DATA_TYPES(ITYPE, DTYPE, NAME, ...)                             \
  [&] {                                                                                 \
    const at::Type& itype = ITYPE;                                                      \
    switch (itype.scalarType()) {                                                       \
      AT_CASE_ITYPE(at::ScalarType::Int, int32_t, DTYPE, NAME, __VA_ARGS__)             \
      AT_CASE_ITYPE(at::ScalarType::Long, int64_t, DTYPE, NAME, __VA_ARGS__)            \
      default:                                                                          \
        AT_ERROR(#NAME, " not implemented for '", itype.toString(), "'");               \
    }                                                                                   \
  }()

namespace {
/*
 * CUDA Kernel of the forward function for Masked Matrix Multiplication:
 * y = adj * (A @ B^T)
 * This is an unoptimized version, to better utilize shared memory, some sort of padding is required.
 * Note that we use the row and col vector to represent the sparse matrix adj. (coo format)
 */
template <typename idx_t, typename data_t>
__global__ void maskedmm_forward_kernel(const idx_t* __restrict__ row, const idx_t* __restrict__ col, const data_t* __restrict__ A, const data_t* __restrict__ Bt, data_t* __restrict__ y, const int e, const int d, const int n, const int h) {
    int i = (((blockIdx.x) * blockDim.x) + (threadIdx.x));
    if (i < e) {
        for (int ko = 0; ko < h; ++ko) {
            data_t sum = 0;
            for (int k = 0; k < d; ++k) {
                sum += A[(row[i] * h + ko) * d + k] * Bt[col[i] + ((ko * d + k) * n)];
            }
            y[i * h + ko] = sum;
        }
    }
}

/*
 * CUDA Kernel of the backward function for Masked Matrix Multiplication: 
 * dA = B @ (dy * adj)
 * dB = A @ (dy * adj)
 * Mostly the same as src_mul_edge
 */
template <typename idx_t, typename data_t>
__global__ void maskedmm_backward_kernel(const idx_t* __restrict__ row, const idx_t* __restrict__ col, const data_t* __restrict__ A, const data_t* __restrict__ B, const data_t* __restrict__ dy, data_t* __restrict__ dA, data_t* __restrict__ dB, const int e, const int d, const int n, const int h) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < d * h) {
        for (int k = 0; k < n; ++k) {
            dA[k * d * h + j] = 0;
            dB[k * d * h + j] = 0;
        }
        for (int k = 0; k < e; ++k) {
            dA[row[k] * d * h + j] += dy[k * h + j / d] * B[col[k] * d * h + j];
            dB[col[k] * d * h + j] += dy[k * h + j / d] * A[row[k] * d * h + j];
        }
    }
}

/*
 * CUDA Kernel of the forward function for Masked Matrix Multiplication. (argument: csr format)
 */
template <typename idx_t, typename data_t>
__global__ void maskedmm_csr_forward_kernel(const idx_t* __restrict__ ptr, const idx_t* __restrict__ eid, const idx_t* __restrict__ nid, const data_t* __restrict__ A, const data_t* __restrict__ B, data_t* __restrict__ y, const int d, const int n, const int h) {
    int i = blockIdx.x; 
    int tx = threadIdx.x;
    if (i < n) {
        for (int j = ptr[i] + tx; j < ptr[i + 1]; j += blockDim.x)
            for (int ko = 0; ko < h; ++ko) {
                data_t sum = 0;
                for (int ki = 0; ki < d; ++ki) {
                    sum += A[(i * h + ko) * d + ki] * B[(ko * d + ki) * n + nid[j]];
                }
                y[eid[j] * h + ko] = sum;
            }
    }
}


/*
 * CUDA Kernel of the backward function for Masked Matrix Multiplication. (argument: csr format)
 */
template <typename idx_t, typename data_t>
__global__ void maskedmm_csr_backward_kernel(const idx_t* __restrict__ ptr_r, const idx_t* __restrict__ eid_r, const idx_t* __restrict__ nid_r, const idx_t* __restrict__ ptr_c, const idx_t* __restrict__ eid_c, const idx_t* __restrict__ nid_c, const data_t* __restrict__ A, const data_t* __restrict__ B, const data_t* __restrict__ dy, data_t* __restrict__ dA, data_t* __restrict__ dB, const int d, const int n, const int h) {
    int tx = threadIdx.x;
    int i = blockIdx.x;
    if (i < n) {
        for (int j = tx; j < d * h; j += blockDim.x) {
            data_t sum = 0;
            for (int k = ptr_r[i]; k < ptr_r[i + 1]; ++k)
                sum += dy[eid_r[k] * h + j / d] * B[nid_r[k] * d * h + j];
            dA[i * d * h + j] = sum;

            sum = 0;
            for (int k = ptr_c[i]; k < ptr_c[i + 1]; ++k)
                sum += dy[eid_c[k] * h + j / d] * A[nid_c[k] * d * h + j];
            dB[i * d * h + j] = sum;
        }
    }
}

/*
 * CUDA Kernel of the forward function for Source Multiply Edge Function.
 * For `src_mul_edge` operation, the arguments are csr(column-major) representations.
 */
template <typename idx_t, typename data_t>
__global__ void vector_spmm_forward_kernel(const idx_t* __restrict__ ptr, const idx_t* __restrict__ eid, const idx_t* __restrict__ nid, const data_t* __restrict__ edata, const data_t* __restrict__ x, data_t* __restrict__ y, const int d, const int n, const int h) {
    int i = blockIdx.x;
    int tx = threadIdx.x;
    if (i < n) {
        for (int j = tx; j < d * h; j += blockDim.x) {
            data_t sum = 0;
            for (int k = ptr[i]; k < ptr[i + 1]; ++k)
                sum += edata[eid[k] * h + j / d] * x[nid[k] * d * h + j];
            y[i * d * h + j] = sum;
        }
    }
}

/*
 * CUDA Kernel of the backward function for Source Multiply Edge Function.
 */
template <typename idx_t, typename data_t>
__global__ void vector_spmm_backward_kernel_0(const idx_t* __restrict__ ptr, const idx_t* __restrict__ eid, const idx_t* __restrict__ nid, const data_t* __restrict__ dy, const data_t* __restrict__ xt, data_t* __restrict__ dedata, const int d, const int n, const int h) {
    int i = blockIdx.x; 
    int tx = threadIdx.x;
    if (i < n) {
        for (int j = ptr[i] + tx; j < ptr[i + 1]; j += blockDim.x)
            for (int ko = 0; ko < h; ++ko) {
                data_t sum = 0;
                for (int ki = 0; ki < d; ++ki) {
                    sum += dy[(i * h + ko) * d + ki] * xt[(ko * d + ki) * n + nid[j]];
                }
                dedata[eid[j] * h + ko] = sum;
            }
    }
}

template <typename idx_t, typename data_t>
__global__ void vector_spmm_backward_kernel_1(const idx_t* __restrict__ ptr, const idx_t* __restrict__ eid, const idx_t* __restrict__ nid, const data_t* __restrict__ edata, const data_t* __restrict__ dy, data_t* __restrict__ dx, const int d, const int n, const int h) {
    int i = blockIdx.x; 
    int tx = threadIdx.x;
    if (i < n) {
        for (int j = tx; j < d * h; j += blockDim.x) {
            data_t sum = 0;
            for (int k = ptr[i]; k < ptr[i + 1]; ++k)
                sum += edata[eid[k] * h + j / d] * dy[nid[k] * d * h + j];
            dx[i * d * h + j] = sum;
        }
    }
}

/*
 * CUDA Kernel of forward function for Sparse Softmax
 * y = softmax(x), grouped by node.
 * ptr, eid: csr format
 */
template <typename idx_t, typename data_t>
__global__ void sparse_softmax_forward_kernel(const idx_t* __restrict__ ptr, const idx_t* __restrict__ eid, const data_t* __restrict__ x, data_t* __restrict__ y, const int n, const int h) {
    //data_t max_val = -1e9;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = threadIdx.y;
    if (i < n) {
        data_t max_val = (ptr[i] < ptr[i + 1]) ? x[eid[ptr[i]] * h + j]: 0;
        for (int k = ptr[i]; k < ptr[i + 1]; ++k)
            max_val = max(max_val, x[eid[k] * h + j]);

        data_t sum = 0;
        for (int k = ptr[i]; k < ptr[i + 1]; ++k) {
            data_t now = exp(x[eid[k] * h + j] - max_val);
            y[eid[k] * h + j] = now;
            sum += now;
        }

        for (int k = ptr[i]; k < ptr[i + 1]; ++k)
            y[eid[k] * h + j] /= sum;
    }
}

/*
 * CUDA Kernel of backward function for Sparse Softmax.
 * ptr, eid: csr format
 */
template <typename idx_t, typename data_t>
__global__ void sparse_softmax_backward_kernel(const idx_t* __restrict__ ptr, const idx_t* __restrict__ eid, const data_t* __restrict__ dy, const data_t* __restrict__ y, data_t* __restrict__ dx, const int n, const int h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    if (i < n) {
        for (int kj = ptr[i] + ty; kj < ptr[i + 1]; kj += blockDim.y) {
            data_t dsum = 0;
            for (int ki = ptr[i]; ki < ptr[i + 1]; ++ki) {
                dsum -= dy[eid[ki] * h + tz] * y[eid[ki] * h + tz] * y[eid[kj] * h + tz];
                if (ki == kj) dsum += dy[eid[ki] * h + tz] * y[eid[ki] * h + tz];
            }
            dx[eid[kj] * h + tz] = dsum;
        }
    }
}

} // End of namespace

at::Tensor maskedmm_cuda_forward(
    const at::Tensor& row,
    const at::Tensor& col,
    const at::Tensor& A,
    const at::Tensor& B) {
    // row, col: (e); A, B: (n, d) or (n, h, d); y: (e, h)
    const auto e = row.size(0);
    const auto n = A.size(0);
    const auto d = A.size(-1);
    const auto h = (A.dim() == 2) ? 1: A.size(1);
    auto y = (h == 1) ? at::zeros({e}, A.options()): at::zeros({e, h}, A.options());

    const int threads = 1024;
    const dim3 blocks((e + threads - 1) / threads);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    auto Bt = (h == 1) ? B.transpose(0, 1).contiguous(): B.permute({1, 2, 0}).contiguous();

    AT_DISPATCH_IDX_DATA_TYPES(row.type(), A.type(), "maskedmm_cuda_forward", ([&] {
        maskedmm_forward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            row.data<idx_t>(),
            col.data<idx_t>(),
            A.data<data_t>(),
            Bt.data<data_t>(),
            y.data<data_t>(),
            e, d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return y;
}

std::vector<at::Tensor> maskedmm_cuda_backward(
    const at::Tensor& row,
    const at::Tensor& col,
    const at::Tensor& A,
    const at::Tensor& B,
    const at::Tensor& dy) {
    // row, col: (e); dy: (e) or (e, h); A, B: (n, d) or (n, h, d);
    const auto e = row.size(0);
    const auto n = A.size(0);
    const auto d = A.size(-1);
    const auto h = (dy.dim() == 2) ? dy.size(1): 1;

    const int threads = 1024; 
    const dim3 blocks((d + threads - 1) / threads);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    auto dA = at::zeros_like(A, A.options());
    auto dB = at::zeros_like(B, B.options());

    AT_DISPATCH_IDX_DATA_TYPES(row.type(), A.type(), "maskedmm_cuda_backward", ([&] {
        maskedmm_backward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            row.data<idx_t>(),
            col.data<idx_t>(),
            A.data<data_t>(),
            B.data<data_t>(),
            dy.data<data_t>(),
            dA.data<data_t>(),
            dB.data<data_t>(),
            e, d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return {dA, dB};
}

// __global__ void maskedmm_csr_forward_kernel(idx_t* __restrict__ ptr, idx_t* __restrict__ eid, idx_t* __restrict__ nid, data_t* __restrict__ A, data_t* __restrict__ B, data_t* __restrict__ y, int d, int n) {
at::Tensor maskedmm_csr_cuda_forward(
    const at::Tensor& ptr,
    const at::Tensor& eid,
    const at::Tensor& nid,
    const at::Tensor& A,
    const at::Tensor& B) {
    // ptr: (n + 1); eid, nid: (e); A, B: (n, d) or (n, h, d); 
    const auto e = eid.size(0);
    const auto n = A.size(0);
    assert(n == ptr.size(0) - 1);
    const auto d = A.size(-1);
    const auto h = (A.dim() == 2) ? 1: A.size(1);
    auto y = (h == 1) ? at::zeros({e}, A.options()): at::zeros({e, h}, A.options());

    const int threads = 32;
    const dim3 blocks(n);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    auto Bt = (B.dim() == 2) ? B.transpose(0, 1).contiguous(): B.permute({1, 2, 0}).contiguous();

    AT_DISPATCH_IDX_DATA_TYPES(ptr.type(), A.type(), "maskedmm_csr_cuda_forward", ([&] {
        maskedmm_csr_forward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            ptr.data<idx_t>(),
            eid.data<idx_t>(),
            nid.data<idx_t>(),
            A.data<data_t>(),
            Bt.data<data_t>(),
            y.data<data_t>(),
            d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return y;
}


// __global__ void maskedmm_csr_backward_kernel(idx_t* __restrict__ ptr_r, idx_t* __restrict__ eid_r, idx_t* __restrict__ nid_r, idx_t* __restrict__ ptr_c, idx_t* __restrict__ eid_c, idx_t* __restrict__ nid_c, data_t* __restrict__ A, data_t* __restrict__ B, data_t* __restrict__ dy, data_t* __restrict__ dA, data_t* __restrict__ dB, int d, int n)
std::vector<at::Tensor> maskedmm_csr_cuda_backward(
    const at::Tensor& ptr_r,
    const at::Tensor& eid_r,
    const at::Tensor& nid_r,
    const at::Tensor& ptr_c,
    const at::Tensor& eid_c,
    const at::Tensor& nid_c,
    const at::Tensor& A,
    const at::Tensor& B,
    const at::Tensor& dy) {
    // ptr_r, ptr_c: (n + 1); eid_r, eid_c, nid_r, eid_c: (e); dy: (e) or (e, h); A, B: (n, d) or (n, h, d)
    const auto e = eid_r.size(0);
    const auto n = A.size(0);
    const auto d = A.size(-1);
    const auto h = (dy.dim() == 2) ? dy.size(1): 1;

    const int threads = 128;
    const dim3 blocks(n);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    auto dA = at::zeros_like(A, A.options());
    auto dB = at::zeros_like(B, B.options());

    AT_DISPATCH_IDX_DATA_TYPES(ptr_r.type(), A.type(), "maskedmm_csr_cuda_backward", ([&] {
        maskedmm_csr_backward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            ptr_r.data<idx_t>(),
            eid_r.data<idx_t>(),
            nid_r.data<idx_t>(),
            ptr_c.data<idx_t>(),
            eid_c.data<idx_t>(),
            nid_c.data<idx_t>(),
            A.data<data_t>(),
            B.data<data_t>(),
            dy.data<data_t>(),
            dA.data<data_t>(),
            dB.data<data_t>(),
            d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return {dA, dB};
}

at::Tensor sparse_softmax_cuda_forward(
    const at::Tensor& ptr,
    const at::Tensor& eid,
    const at::Tensor& x) {
    // ptr: (n + 1); eid: (e); x: (e) or (e, h);
    const auto n = ptr.size(0) - 1;
    const auto h = (x.dim() == 2) ? x.size(1): 1;
    assert(h <= 32);
    const dim3 threads(32, h);
    const dim3 blocks((n + threads.x - 1) / threads.x);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    const auto y = at::zeros_like(x, x.options());
    
    AT_DISPATCH_IDX_DATA_TYPES(eid.type(), x.type(), "sparse_softmax_cuda_forward",([&] {
        sparse_softmax_forward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            ptr.data<idx_t>(),
            eid.data<idx_t>(),
            x.data<data_t>(),
            y.data<data_t>(),
            n, h);
    }));
    THCudaCheck(hipGetLastError());
    return y;
}

at::Tensor sparse_softmax_cuda_backward(
    const at::Tensor& ptr,
    const at::Tensor& eid,
    const at::Tensor& y,
    const at::Tensor& dy) {
    // ptr: (n + 1); eid: (e); y: (e) or (e, h); dy: (e) or (e, h);
    const auto n = ptr.size(0) - 1;
    const auto h = (dy.dim() == 2) ? dy.size(1): 1;
    assert(h <= 32);
    const dim3 threads(1, 32, h);
    const dim3 blocks((n + threads.x - 1) / threads.x);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    const auto dx = at::zeros_like(dy, dy.options());

    AT_DISPATCH_IDX_DATA_TYPES(eid.type(), y.type(), "sparse_softmax_cuda_backward", ([&] {
        sparse_softmax_backward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            ptr.data<idx_t>(),
            eid.data<idx_t>(),
            dy.data<data_t>(),
            y.data<data_t>(),
            dx.data<data_t>(),
            n, h); 
    }));
    THCudaCheck(hipGetLastError());
    return dx;
}

at::Tensor vector_spmm_cuda_forward(
    const at::Tensor& ptr,
    const at::Tensor& eid,
    const at::Tensor& nid,
    const at::Tensor& edata,
    const at::Tensor& x) {
    // ptr: (n + 1); eid, nid: (e); edata: (e) or (e, h); x: (n, d) or (n, h, d);
    const auto n = ptr.size(0) - 1;
    const auto h = (edata.dim() == 2) ? edata.size(1): 1;
    const auto d = x.size(-1); 
    
    const int threads = 128;
    const dim3 blocks(n);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const auto y = at::zeros_like(x, x.options());
    
    AT_DISPATCH_IDX_DATA_TYPES(eid.type(), x.type(), "vector_spmm_forward", ([&] {
        vector_spmm_forward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            ptr.data<idx_t>(),
            eid.data<idx_t>(),
            nid.data<idx_t>(),
            edata.data<data_t>(),
            x.data<data_t>(),
            y.data<data_t>(),
            d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return y;
}

std::vector<at::Tensor> vector_spmm_cuda_backward(
    const at::Tensor& ptr,
    const at::Tensor& eid,
    const at::Tensor& nid,
    const at::Tensor& ptr_t,
    const at::Tensor& eid_t,
    const at::Tensor& nid_t,
    const at::Tensor& edata,
    const at::Tensor& dy,
    const at::Tensor& x) {
    // ptr: (n + 1); eid, nid: (e); edata: (e) or (e, h); dy, x: (n, d) or (n, h, d); 
    const auto n = ptr.size(0) - 1;
    const auto h = (edata.dim() == 2) ? edata.size(1): 1;
    const auto d = x.size(-1);

    int threads = 32;
    const dim3 blocks(n);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    const auto xt = (h == 1) ? x.transpose(0, 1).contiguous(): x.permute({1, 2, 0}).contiguous();

    const auto dx = at::zeros_like(x, x.options());
    const auto dedata = at::zeros_like(edata, edata.options());

    AT_DISPATCH_IDX_DATA_TYPES(eid.type(), x.type(), "vector_spmm_backward_0", ([&] {
        vector_spmm_backward_kernel_0<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            ptr.data<idx_t>(),
            eid.data<idx_t>(),
            nid.data<idx_t>(),
            dy.data<data_t>(),
            xt.data<data_t>(),
            dedata.data<data_t>(),
            d, n, h);
    }));

    threads = 128;
    
    AT_DISPATCH_IDX_DATA_TYPES(eid.type(), x.type(), "vector_spmm_backward_1", ([&] {
        vector_spmm_backward_kernel_1<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            ptr_t.data<idx_t>(),
            eid_t.data<idx_t>(),
            nid_t.data<idx_t>(),
            edata.data<data_t>(),
            dy.data<data_t>(),
            dx.data<data_t>(),
            d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return {dedata, dx};
}
