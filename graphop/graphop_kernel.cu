#include "hip/hip_runtime.h"
/* TODOs
 * - segment_reduce_forward, segment_reduce_backward;
 * - switch backend from aten to dlpack
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Type.h>
#include <c10/util/Exception.h>
#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define AT_CASE_ITYPE(enum_type, type, DTYPE, NAME, ...)                    \
  case enum_type: {                                                         \
    const at::Type& dtype = DTYPE;                                          \
    using idx_t = type;                                                     \
    switch (dtype.scalarType()) {                                           \
      case at::ScalarType::Half: {                                          \
        using data_t = at::Half;                                            \
        return __VA_ARGS__();                                               \
      }                                                                     \
      case at::ScalarType::Float: {                                         \
        using data_t = float;                                               \
        return __VA_ARGS__();                                               \
      }                                                                     \
      case at::ScalarType::Double: {                                        \
        using data_t = double;                                              \
        return __VA_ARGS__();                                               \
      }                                                                     \
      default:                                                              \
        AT_ERROR(#NAME, " not implemented for '", dtype.toString(), "'");   \
    }                                                                       \
  }                

#define AT_DISPATCH_IDX_DATA_TYPES(ITYPE, DTYPE, NAME, ...)                             \
  [&] {                                                                                 \
    const at::Type& itype = ITYPE;                                                      \
    switch (itype.scalarType()) {                                                       \
      AT_CASE_ITYPE(at::ScalarType::Int, int32_t, DTYPE, NAME, __VA_ARGS__)             \
      AT_CASE_ITYPE(at::ScalarType::Long, int64_t, DTYPE, NAME, __VA_ARGS__)            \
      default:                                                                          \
        AT_ERROR(#NAME, " not implemented for '", itype.toString(), "'");               \
    }                                                                                   \
  }()

namespace {

/*
 * CUDA Kernel of the forward function for Masked Matrix Multiplication:
 * y = adj * (A @ B^T)
 * This is an unoptimized version, to better utilize shared memory, some sort of padding is required.
 * Note that we use the row and col vector to represent the sparse matrix adj. (coo format)
 */
template <typename idx_t, typename data_t>
__global__ void maskedmm_forward_kernel(const idx_t* __restrict__ row, const idx_t* __restrict__ col, const data_t* __restrict__ A, const data_t* __restrict__ Bt, data_t* __restrict__ y, const int e, const int d, const int n, const int h) {
    int i = (((blockIdx.x) * blockDim.x) + (threadIdx.x));
    if (i < e) {
        for (int ko = 0; ko < h; ++ko) {
            data_t sum = 0;
            for (int k = 0; k < d; ++k) {
                sum += A[(row[i] * h + ko) * d + k] * Bt[col[i] + ((ko * d + k) * n)];
            }
            y[i * h + ko] = sum;
        }
    }
}

/*
 * CUDA Kernel of the backward function for Masked Matrix Multiplication: 
 * dA = B @ (dy * adj)
 * dB = A @ (dy * adj)
 * Mostly the same as src_mul_edge
 */
template <typename idx_t, typename data_t>
__global__ void maskedmm_backward_kernel(const idx_t* __restrict__ row, const idx_t* __restrict__ col, const data_t* __restrict__ A, const data_t* __restrict__ B, const data_t* __restrict__ dy, data_t* __restrict__ dA, data_t* __restrict__ dB, const int e, const int d, const int n, const int h) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < d * h) {
        for (int k = 0; k < n; ++k) {
            dA[k * d * h + j] = 0;
            dB[k * d * h + j] = 0;
        }
        for (int k = 0; k < e; ++k) {
            dA[row[k] * d * h + j] += dy[k * h + j / d] * B[col[k] * d * h + j];
            dB[col[k] * d * h + j] += dy[k * h + j / d] * A[row[k] * d * h + j];
        }
    }
}

/*
 * CUDA Kernel of the forward function for Node-Edge Multiplication(reduced on edge, designed for relative positional encoding).
 */
template <typename idx_t, typename data_t>
__global__ void node_mul_edge_forward_kernel(const idx_t* __restrict__ indptr, const idx_t* __restrict__ eid, const data_t* __restrict__ A, const data_t* __restrict__ B, data_t* __restrict__ y, const int d, const int n, const int h) {
    int i = blockIdx.x;
    int tx = threadIdx.x;
    if (i < n) {
        for (int j = indptr[i] + tx; j < indptr[i + 1]; j += blockDim.x)
            for (int ko = 0; ko < h; ++ko) {
                data_t sum = 0;
                for (int ki = 0; ki < d; ++ki) {
                    sum += A[(i * h + ko) * d + ki] * B[eid[j] * d + ki];
                }
                y[eid[j] * h + ko] = sum;
            }
    }
}


/*
 * CUDA Kernel of the forward function for Masked Matrix Multiplication. (argument: csr format)
 */
template <typename idx_t, typename data_t>
__global__ void maskedmm_csr_forward_kernel(const idx_t* __restrict__ indptr, const idx_t* __restrict__ eid, const idx_t* __restrict__ indices, const data_t* __restrict__ A, const data_t* __restrict__ B, data_t* __restrict__ y, const int d, const int n, const int h) {
    int i = blockIdx.x; 
    int tx = threadIdx.x;
    if (i < n) {
        for (int j = indptr[i] + tx; j < indptr[i + 1]; j += blockDim.x)
            for (int ko = 0; ko < h; ++ko) {
                data_t sum = 0;
                for (int ki = 0; ki < d; ++ki) {
                    sum += A[(i * h + ko) * d + ki] * B[(ko * d + ki) * n + indices[j]];
                }
                y[eid[j] * h + ko] = sum;
            }
    }
}


/*
 * CUDA Kernel of the backward function for Node-Edge Multiplication(reduced on edge, designed for relative positional encoding).
 */
template <typename idx_t, typename data_t>
__global__ void node_mul_edge_backward_kernel_0(const idx_t* __restrict__ indptr, const idx_t* __restrict__ eid, const data_t* __restrict__ B, const data_t* __restrict__ dy, data_t* __restrict__ dA, const int d, const int n, const int h) {
    int tx = threadIdx.x;
    int i = blockIdx.x;
    if (i < n) {
        for (int j = tx; j < d * h; j += blockDim.x) {
            data_t sum = 0;
            for (int k = indptr[i]; k < indptr[i + 1]; ++k)
                sum += dy[eid[k] * h + j / d] * B[eid[k] * d + j % d];
            dA[i * d * h + j] = sum; 
        }
    }
}


/*
 * CUDA Kernel of the backward function for Node-Edge Multiplication(reduced on edge, designed for relative positional encoding).
 */
template <typename idx_t, typename data_t>
__global__ void node_mul_edge_backward_kernel_1(const idx_t* __restrict__ indptr, const idx_t* __restrict__ eid, const data_t* __restrict__ A, const data_t* __restrict__ dy, data_t* __restrict__ dB, const int d, const int n, const int h) {
    int tx = threadIdx.x;
    int i = blockIdx.x;
    if (i < n) {
        for (int j = tx; j < d; j += blockDim.x) {
            for (int k = indptr[i]; k < indptr[i + 1]; ++k) {
                data_t sum = 0;
                for (int ki = 0; ki < h; ++ki) {
                    sum += dy[eid[k] * h + ki] * A[(i * h + ki) * d + j];
                }
                dB[eid[k] * d + j] = sum;
            }
        }
    }
}


/*
 * CUDA Kernel of the backward function for Masked Matrix Multiplication. (argument: csr format)
 */
template <typename idx_t, typename data_t>
__global__ void maskedmm_csr_backward_kernel(const idx_t* __restrict__ indptr_r, const idx_t* __restrict__ eid_r, const idx_t* __restrict__ indices_r, const idx_t* __restrict__ indptr_c, const idx_t* __restrict__ eid_c, const idx_t* __restrict__ indices_c, const data_t* __restrict__ A, const data_t* __restrict__ B, const data_t* __restrict__ dy, data_t* __restrict__ dA, data_t* __restrict__ dB, const int d, const int n, const int h) {
    int tx = threadIdx.x;
    int i = blockIdx.x;
    if (i < n) {
        for (int j = tx; j < d * h; j += blockDim.x) {
            data_t sum = 0;
            for (int k = indptr_r[i]; k < indptr_r[i + 1]; ++k)
                sum += dy[eid_r[k] * h + j / d] * B[indices_r[k] * d * h + j];
            dA[i * d * h + j] = sum;

            sum = 0;
            for (int k = indptr_c[i]; k < indptr_c[i + 1]; ++k)
                sum += dy[eid_c[k] * h + j / d] * A[indices_c[k] * d * h + j];
            dB[i * d * h + j] = sum;
        }
    }
}

/*
 * CUDA Kernel of the forward function for Source Multiply Edge Function.
 * For `src_mul_edge` operation, the arguments are csr(column-major) representations.
 */
template <typename idx_t, typename data_t>
__global__ void vector_spmm_forward_kernel(const idx_t* __restrict__ indptr, const idx_t* __restrict__ eid, const idx_t* __restrict__ indices, const data_t* __restrict__ edata, const data_t* __restrict__ x, data_t* __restrict__ y, const int d, const int n, const int h) {
    int i = blockIdx.x;
    int tx = threadIdx.x;
    if (i < n) {
        for (int j = tx; j < d * h; j += blockDim.x) {
            data_t sum = 0;
            for (int k = indptr[i]; k < indptr[i + 1]; ++k)
                sum += edata[eid[k] * h + j / d] * x[indices[k] * d * h + j];
            y[i * d * h + j] = sum;
        }
    }
}

/*
 * CUDA Kernel of the backward function for Source Multiply Edge Function.
 */
template <typename idx_t, typename data_t>
__global__ void vector_spmm_backward_kernel_0(const idx_t* __restrict__ indptr, const idx_t* __restrict__ eid, const idx_t* __restrict__ indices, const data_t* __restrict__ dy, const data_t* __restrict__ xt, data_t* __restrict__ dedata, const int d, const int n, const int h) {
    int i = blockIdx.x; 
    int tx = threadIdx.x;
    if (i < n) {
        for (int j = indptr[i] + tx; j < indptr[i + 1]; j += blockDim.x)
            for (int ko = 0; ko < h; ++ko) {
                data_t sum = 0;
                for (int ki = 0; ki < d; ++ki) {
                    sum += dy[(i * h + ko) * d + ki] * xt[(ko * d + ki) * n + indices[j]];
                }
                dedata[eid[j] * h + ko] = sum;
            }
    }
}

template <typename idx_t, typename data_t>
__global__ void vector_spmm_backward_kernel_1(const idx_t* __restrict__ indptr, const idx_t* __restrict__ eid, const idx_t* __restrict__ indices, const data_t* __restrict__ edata, const data_t* __restrict__ dy, data_t* __restrict__ dx, const int d, const int n, const int h) {
    int i = blockIdx.x; 
    int tx = threadIdx.x;
    if (i < n) {
        for (int j = tx; j < d * h; j += blockDim.x) {
            data_t sum = 0;
            for (int k = indptr[i]; k < indptr[i + 1]; ++k)
                sum += edata[eid[k] * h + j / d] * dy[indices[k] * d * h + j];
            dx[i * d * h + j] = sum;
        }
    }
}

/*
 * CUDA Kernel of forward function for Sparse Softmax
 * y = softmax(x), grouped by node.
 * indptr, eid: csr format
 */
template <typename idx_t, typename data_t>
__global__ void sparse_softmax_forward_kernel(const idx_t* __restrict__ indptr, const idx_t* __restrict__ eid, const data_t* __restrict__ x, data_t* __restrict__ y, const int n, const int h) {
    //data_t max_val = -1e9;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = threadIdx.y;
    if (i < n) {
        data_t max_val = (indptr[i] < indptr[i + 1]) ? x[eid[indptr[i]] * h + j]: (data_t)(0);
        for (int k = indptr[i]; k < indptr[i + 1]; ++k)
            max_val = max(max_val, x[eid[k] * h + j]);

        data_t sum = 0;
        for (int k = indptr[i]; k < indptr[i + 1]; ++k) {
            data_t now = exp(x[eid[k] * h + j] - max_val);
            y[eid[k] * h + j] = now;
            sum += now;
        }

        for (int k = indptr[i]; k < indptr[i + 1]; ++k)
            y[eid[k] * h + j] /= sum;
    }
}

/*
 * CUDA Kernel of backward function for Sparse Softmax.
 * indptr, eid: csr format
 */
template <typename idx_t, typename data_t>
__global__ void sparse_softmax_backward_kernel(const idx_t* __restrict__ indptr, const idx_t* __restrict__ eid, const data_t* __restrict__ dy, const data_t* __restrict__ y, data_t* __restrict__ dx, const int n, const int h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    if (i < n) {
        for (int kj = indptr[i] + ty; kj < indptr[i + 1]; kj += blockDim.y) {
            data_t dsum = 0;
            for (int ki = indptr[i]; ki < indptr[i + 1]; ++ki) {
                dsum -= dy[eid[ki] * h + tz] * y[eid[ki] * h + tz] * y[eid[kj] * h + tz];
                if (ki == kj) dsum += dy[eid[ki] * h + tz] * y[eid[ki] * h + tz];
            }
            dx[eid[kj] * h + tz] = dsum;
        }
    }
}

} // End of namespace

at::Tensor maskedmm_cuda_forward(
    const at::Tensor& row,
    const at::Tensor& col,
    const at::Tensor& A,
    const at::Tensor& B) {
    // row, col: (e); A, B: (n, d) or (n, h, d); y: (e, h)
    hipSetDevice(row.get_device());

    const auto e = row.size(0);
    const auto n = A.size(0);
    const auto d = A.size(-1);
    const auto h = (A.dim() == 2) ? 1: A.size(1);
    auto y = (h == 1) ? at::zeros({e}, A.options()): at::zeros({e, h}, A.options());

    const int threads = 1024;
    const dim3 blocks((e + threads - 1) / threads);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    auto Bt = (h == 1) ? B.transpose(0, 1).contiguous(): B.permute({1, 2, 0}).contiguous();

    AT_DISPATCH_IDX_DATA_TYPES(row.type(), A.type(), "maskedmm_cuda_forward", ([&] {
        maskedmm_forward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            row.data<idx_t>(),
            col.data<idx_t>(),
            A.data<data_t>(),
            Bt.data<data_t>(),
            y.data<data_t>(),
            e, d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return y;
}

std::vector<at::Tensor> maskedmm_cuda_backward(
    const at::Tensor& row,
    const at::Tensor& col,
    const at::Tensor& A,
    const at::Tensor& B,
    const at::Tensor& dy) {
    // row, col: (e); dy: (e) or (e, h); A, B: (n, d) or (n, h, d);
    hipSetDevice(row.get_device());

    const auto e = row.size(0);
    const auto n = A.size(0);
    const auto d = A.size(-1);
    const auto h = (dy.dim() == 2) ? dy.size(1): 1;

    const int threads = 1024; 
    const dim3 blocks((d + threads - 1) / threads);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    auto dA = at::zeros_like(A, A.options());
    auto dB = at::zeros_like(B, B.options());

    AT_DISPATCH_IDX_DATA_TYPES(row.type(), A.type(), "maskedmm_cuda_backward", ([&] {
        maskedmm_backward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            row.data<idx_t>(),
            col.data<idx_t>(),
            A.data<data_t>(),
            B.data<data_t>(),
            dy.data<data_t>(),
            dA.data<data_t>(),
            dB.data<data_t>(),
            e, d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return {dA, dB};
}

at::Tensor node_mul_edge_cuda_forward(
    const at::Tensor& indptr,
    const at::Tensor& eid,
    const at::Tensor& A,
    const at::Tensor& B) {
    // indptr: (n + 1); eid: (e); A: (n, d) or (n, h, d); B: (e, d);
    hipSetDevice(indptr.get_device());

    const auto e = eid.size(0);
    const auto n = A.size(0);
    assert(n == indptr.size(0) - 1);
    const auto d = A.size(-1);
    const auto h = (A.dim() == 2) ? 1: A.size(1);
    auto y = (h == 1) ? at::zeros({e}, A.options()): at::zeros({e, h}, A.options());

    const int threads = 32;
    const dim3 blocks(n);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_IDX_DATA_TYPES(indptr.type(), A.type(), "node_mul_edge_cuda_forward", ([&] {
        node_mul_edge_forward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            indptr.data<idx_t>(),
            eid.data<idx_t>(),
            A.data<data_t>(),
            B.data<data_t>(),
            y.data<data_t>(),
            d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return y;
}

// __global__ void maskedmm_csr_forward_kernel(idx_t* __restrict__ indptr, idx_t* __restrict__ eid, idx_t* __restrict__ indices, data_t* __restrict__ A, data_t* __restrict__ B, data_t* __restrict__ y, int d, int n) {
at::Tensor maskedmm_csr_cuda_forward(
    const at::Tensor& indptr,
    const at::Tensor& eid,
    const at::Tensor& indices,
    const at::Tensor& A,
    const at::Tensor& B) {
    // indptr: (n + 1); eid, indices: (e); A, B: (n, d) or (n, h, d); 
    hipSetDevice(indptr.get_device());

    const auto e = eid.size(0);
    const auto n = A.size(0);
    assert(n == indptr.size(0) - 1);
    const auto d = A.size(-1);
    const auto h = (A.dim() == 2) ? 1: A.size(1);
    auto y = (h == 1) ? at::zeros({e}, A.options()): at::zeros({e, h}, A.options());

    const int threads = 32;
    const dim3 blocks(n);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    auto Bt = (B.dim() == 2) ? B.transpose(0, 1).contiguous(): B.permute({1, 2, 0}).contiguous();

    AT_DISPATCH_IDX_DATA_TYPES(indptr.type(), A.type(), "maskedmm_csr_cuda_forward", ([&] {
        maskedmm_csr_forward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            indptr.data<idx_t>(),
            eid.data<idx_t>(),
            indices.data<idx_t>(),
            A.data<data_t>(),
            Bt.data<data_t>(),
            y.data<data_t>(),
            d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return y;
}

std::vector<at::Tensor> node_mul_edge_cuda_backward(
    const at::Tensor& indptr,
    const at::Tensor& eid,
    const at::Tensor& A,
    const at::Tensor& B,
    const at::Tensor& dy) {
    // indptr: (n + 1); eid: (e); dy: (e) or (e, h); A: (n, d) or (n, h, d); B: (e, d)
    hipSetDevice(indptr.get_device());

    const auto e = eid.size(0);
    const auto n = A.size(0);
    const auto d = A.size(-1);
    const auto h = (dy.dim() == 2) ? dy.size(1): 1;

    int threads = 128;
    const dim3 blocks(n);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    auto dA = at::zeros_like(A, A.options());
    auto dB = at::zeros_like(B, B.options());

    AT_DISPATCH_IDX_DATA_TYPES(indptr.type(), A.type(), "node_mul_edge_cuda_backward_0", ([&] {
        node_mul_edge_backward_kernel_0<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            indptr.data<idx_t>(),
            eid.data<idx_t>(),
            B.data<data_t>(),
            dy.data<data_t>(),
            dA.data<data_t>(),
            d, n, h);
    }));
    threads = d;
    AT_DISPATCH_IDX_DATA_TYPES(indptr.type(), A.type(), "node_mul_edge_cuda_backward_1", ([&] {
        node_mul_edge_backward_kernel_1<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            indptr.data<idx_t>(),
            eid.data<idx_t>(),
            A.data<data_t>(),
            dy.data<data_t>(),
            dB.data<data_t>(),
            d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return {dA, dB};
}


// __global__ void maskedmm_csr_backward_kernel(idx_t* __restrict__ indptr_r, idx_t* __restrict__ eid_r, idx_t* __restrict__ indices_r, idx_t* __restrict__ indptr_c, idx_t* __restrict__ eid_c, idx_t* __restrict__ indices_c, data_t* __restrict__ A, data_t* __restrict__ B, data_t* __restrict__ dy, data_t* __restrict__ dA, data_t* __restrict__ dB, int d, int n)
std::vector<at::Tensor> maskedmm_csr_cuda_backward(
    const at::Tensor& indptr_r,
    const at::Tensor& eid_r,
    const at::Tensor& indices_r,
    const at::Tensor& indptr_c,
    const at::Tensor& eid_c,
    const at::Tensor& indices_c,
    const at::Tensor& A,
    const at::Tensor& B,
    const at::Tensor& dy) {
    // indptr_r, indptr_c: (n + 1); eid_r, eid_c, indices_r, indices_c: (e); dy: (e) or (e, h); A, B: (n, d) or (n, h, d)
    hipSetDevice(indptr_r.get_device());

    const auto e = eid_r.size(0);
    const auto n = A.size(0);
    const auto d = A.size(-1);
    const auto h = (dy.dim() == 2) ? dy.size(1): 1;

    const int threads = 128;
    const dim3 blocks(n);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    auto dA = at::zeros_like(A, A.options());
    auto dB = at::zeros_like(B, B.options());

    AT_DISPATCH_IDX_DATA_TYPES(indptr_r.type(), A.type(), "maskedmm_csr_cuda_backward", ([&] {
        maskedmm_csr_backward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            indptr_r.data<idx_t>(),
            eid_r.data<idx_t>(),
            indices_r.data<idx_t>(),
            indptr_c.data<idx_t>(),
            eid_c.data<idx_t>(),
            indices_c.data<idx_t>(),
            A.data<data_t>(),
            B.data<data_t>(),
            dy.data<data_t>(),
            dA.data<data_t>(),
            dB.data<data_t>(),
            d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return {dA, dB};
}

at::Tensor sparse_softmax_cuda_forward(
    const at::Tensor& indptr,
    const at::Tensor& eid,
    const at::Tensor& x) {
    hipSetDevice(indptr.get_device());

    // indptr: (n + 1); eid: (e); x: (e) or (e, h);
    const auto n = indptr.size(0) - 1;
    const auto h = (x.dim() == 2) ? x.size(1): 1;
    assert(h <= 32);
    const dim3 threads(32, h);
    const dim3 blocks((n + threads.x - 1) / threads.x);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    const auto y = at::zeros_like(x, x.options());
    
    AT_DISPATCH_IDX_DATA_TYPES(eid.type(), x.type(), "sparse_softmax_cuda_forward",([&] {
        sparse_softmax_forward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            indptr.data<idx_t>(),
            eid.data<idx_t>(),
            x.data<data_t>(),
            y.data<data_t>(),
            n, h);
    }));
    THCudaCheck(hipGetLastError());
    return y;
}

at::Tensor sparse_softmax_cuda_backward(
    const at::Tensor& indptr,
    const at::Tensor& eid,
    const at::Tensor& y,
    const at::Tensor& dy) {
    hipSetDevice(indptr.get_device());

    // indptr: (n + 1); eid: (e); y: (e) or (e, h); dy: (e) or (e, h);
    const auto n = indptr.size(0) - 1;
    const auto h = (dy.dim() == 2) ? dy.size(1): 1;
    assert(h <= 32);
    const dim3 threads(1, 32, h);
    const dim3 blocks((n + threads.x - 1) / threads.x);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    const auto dx = at::zeros_like(dy, dy.options());

    AT_DISPATCH_IDX_DATA_TYPES(eid.type(), y.type(), "sparse_softmax_cuda_backward", ([&] {
        sparse_softmax_backward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            indptr.data<idx_t>(),
            eid.data<idx_t>(),
            dy.data<data_t>(),
            y.data<data_t>(),
            dx.data<data_t>(),
            n, h); 
    }));
    THCudaCheck(hipGetLastError());
    return dx;
}

at::Tensor vector_spmm_cuda_forward(
    const at::Tensor& indptr,
    const at::Tensor& eid,
    const at::Tensor& indices,
    const at::Tensor& edata,
    const at::Tensor& x) {
    // indptr: (n + 1); eid, indices: (e); edata: (e) or (e, h); x: (n, d) or (n, h, d);
    hipSetDevice(indptr.get_device());

    const auto n = indptr.size(0) - 1;
    const auto h = (edata.dim() == 2) ? edata.size(1): 1;
    const auto d = x.size(-1); 
    
    const int threads = 128;
    const dim3 blocks(n);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const auto y = at::zeros_like(x, x.options());
    
    AT_DISPATCH_IDX_DATA_TYPES(eid.type(), x.type(), "vector_spmm_forward", ([&] {
        vector_spmm_forward_kernel<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            indptr.data<idx_t>(),
            eid.data<idx_t>(),
            indices.data<idx_t>(),
            edata.data<data_t>(),
            x.data<data_t>(),
            y.data<data_t>(),
            d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return y;
}

std::vector<at::Tensor> vector_spmm_cuda_backward(
    const at::Tensor& indptr,
    const at::Tensor& eid,
    const at::Tensor& indices,
    const at::Tensor& indptr_t,
    const at::Tensor& eid_t,
    const at::Tensor& indices_t,
    const at::Tensor& edata,
    const at::Tensor& dy,
    const at::Tensor& x) {
    // indptr: (n + 1); eid, indices: (e); edata: (e) or (e, h); dy, x: (n, d) or (n, h, d); 
    hipSetDevice(indptr.get_device());

    const auto n = indptr.size(0) - 1;
    const auto h = (edata.dim() == 2) ? edata.size(1): 1;
    const auto d = x.size(-1);

    int threads = 32;
    const dim3 blocks(n);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    const auto xt = (h == 1) ? x.transpose(0, 1).contiguous(): x.permute({1, 2, 0}).contiguous();

    const auto dx = at::zeros_like(x, x.options());
    const auto dedata = at::zeros_like(edata, edata.options());

    AT_DISPATCH_IDX_DATA_TYPES(eid.type(), x.type(), "vector_spmm_backward_0", ([&] {
        vector_spmm_backward_kernel_0<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            indptr.data<idx_t>(),
            eid.data<idx_t>(),
            indices.data<idx_t>(),
            dy.data<data_t>(),
            xt.data<data_t>(),
            dedata.data<data_t>(),
            d, n, h);
    }));

    threads = 128;
    AT_DISPATCH_IDX_DATA_TYPES(eid.type(), x.type(), "vector_spmm_backward_1", ([&] {
        vector_spmm_backward_kernel_1<idx_t, data_t><<<blocks, threads, 0, stream>>>(
            indptr_t.data<idx_t>(),
            eid_t.data<idx_t>(),
            indices_t.data<idx_t>(),
            edata.data<data_t>(),
            dy.data<data_t>(),
            dx.data<data_t>(),
            d, n, h);
    }));
    THCudaCheck(hipGetLastError());
    return {dedata, dx};
}
