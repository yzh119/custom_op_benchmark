#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <vector>

/*
 * CUDA Kernel of the forward function for Masked Matrix Multiplication:
 * O = adj * (A @ B)
 * This is an unoptimized version, to better utilize shared memory, some sort of padding is required.
 * Note that we use the row and col vector to represent the sparse matrix adj.
 */

__global__ void maskedmm_forward_kernel(int* __restrict__ row, int* __restrict__ col, float* __restrict__ A, float* __restrict__ B, float* __restrict__ O, int e, int d, int n) {
    int i = ((((int)blockIdx.x) * (int)blockDim.x) + ((int)threadIdx.x));
    if (((int)blockIdx.x) < (e / (int)blockDim.x)) {
        O[i] = 0.000000e+00f;
        for (int k = 0; k < d; ++k) {
            O[i] = (O[i] + (A[((row[i] * d) + k)] * B[(col[i] + (k * n))]));
        }
    } else {
        if (i < e) {
            O[i] = 0.000000e+00f;
        }
        for (int k = 0; k < d; ++k) {
            if (i < e) {
                O[i] = (O[i] + (A[((row[i] * d) + k)] * B[(col[i] + (k * n))]));
            }
        }
    }
}

/*
 * CUDA Kernel of the backward function for Masked Matrix Multiplication: 
 * dA = B @ (dO * adj)
 * dB = A @ (dO * adj)
 * Mostly the same as src_mul_edge
 */
__global__ void maskedmm_backward_kernel(int* __restrict__ row, int* __restrict__ col, float* __restrict__ A, float* __restrict__ B, float* __restrict__ dO, float* __restrict__ dA, float* __restrict__ dB, int e, int d, int n) {
    int j = (int)blockIdx.x * (int)blockDim.x + (int)threadIdx.x;
    for (int k = 0; k < n; ++k) {
		dA[k * d + j] = 0;
		dB[k * d + j] = 0;
	}

    for (int k = 0; k < e; ++k) {
		dA[row[k] * d + j] += dO[k] * B[col[k] * d + j];
		dB[col[k] * d + j] += dO[k] * A[row[k] * d + j];
    }
}

/*
 * CUDA Kernel of forward function for Sparse Softmax
 * O = softmax(x), grouped by node.
 * head, idx: csr format (row-major)
 */
__global__ void sparse_softmax_forward_kernel(int* __restrict__ head, int* __restrict__ idx, float* __restrict__ x, float* __restrict__ O, int e) {
    float max_val = *x;
    int j = (int)threadIdx.x;
    if (j < e) {
        for (int k = head[j]; k < head[j + 1]; ++k)
            max_val = max(max_val, x[idx[k]]);

	float sum = 0;
	for (int k = head[j]; k < head[j + 1]; ++k) {
		float now = exp(x[idx[k]] - max_val);
		O[idx[k]] = now;
		sum += now;
	}

	for (int k = head[j]; k < head[j + 1]; ++k)
		O[idx[k]] /= sum;
    }
}

/*
 * CUDA Kernel of backward function for Sparse Softmax.
 * head, idx: csr format (col-major)
 */
__global__ void sparse_softmax_backward_kernel(int* __restrict__ head, int* __restrict__ idx, float* __restrict__ dO, float* __restrict__ O, float* __restrict__ dx, int e) {
    int i = (int)blockIdx.x;
    if (i < e) {
        for (int ki = head[i]; ki < head[i + 1]; ++ki) {
            for (int kj = head[i]; kj < head[i + 1]; ++kj) {
                dx[idx[kj]] -= dO[idx[ki]] * O[idx[ki]] * O[idx[kj]];
                if (ki == kj) dx[idx[kj]] += dO[idx[ki]] * O[idx[ki]];
            }
        }
    }
}

at::Tensor maskedmm_cuda_forward(
    at::Tensor row,
    at::Tensor col,
    at::Tensor A,
    at::Tensor B) {
    // row, col: (e), A, B: (n, d), O: (e)
    const auto e = row.size(0);
    const auto n = A.size(0);
    const auto d = A.size(1);
    auto O = at::zeros({e}, A.options());

    const int threads = 32;
    const dim3 blocks((e + threads - 1) / threads);
    auto Bt = B.transpose(0, 1).contiguous();

    maskedmm_forward_kernel<<<blocks, threads>>>(
        row.data<int>(),
        col.data<int>(),
        A.data<float>(),
        Bt.data<float>(),
        O.data<float>(),
        e, d, n);
    return O;
}

std::vector<at::Tensor> maskedmm_cuda_backward(
    at::Tensor row,
    at::Tensor col,
    at::Tensor A,
    at::Tensor B,
    at::Tensor dO) {
    // row, col: (e), dO: (e), A, B: (n, d)
    const auto e = row.size(0);
    const auto n = A.size(0);
    const auto d = A.size(1);

    const int threads = 1024; 
    const dim3 blocks((d + 1023) / 1024);

    auto dA = at::zeros_like(A, A.options());
    auto dB = at::zeros_like(B, B.options());
    dO = dO.contiguous();

    maskedmm_backward_kernel<<<blocks, threads>>>(
        row.data<int>(),
        col.data<int>(),
        A.data<float>(),
        B.data<float>(),
        dO.data<float>(),
        dA.data<float>(),
        dB.data<float>(),
        e, d, n);
    return {dA, dB};
}
